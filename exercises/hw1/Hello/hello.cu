#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){

  hello<<<100, 12>>>();
  hipDeviceSynchronize();
}
